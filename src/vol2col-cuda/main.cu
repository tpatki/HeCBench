#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <algorithm>
#include <chrono>
#include <cmath>
#include <hip/hip_runtime.h>
#include "reference.h"

#define threadsPerBlock 512

// Kernel for fast unfold+copy on volumes
template <typename T>
__global__ void vol2col_kernel(
    const int64_t range,
    const T* data_vol,
    const int depth,
    const int height,
    const int width,
    const int ksize_t,
    const int ksize_h,
    const int ksize_w,
    const int pad_t,
    const int pad_h,
    const int pad_w,
    const int stride_t,
    const int stride_h,
    const int stride_w,
    const int dilation_t,
    const int dilation_h,
    const int dilation_w,
    const int depth_col,
    const int height_col,
    const int width_col,
    T* data_col)
{
  for (int64_t n = blockDim.x * blockIdx.x + threadIdx.x;
               n < range; n += blockDim.x * gridDim.x) {
    int w_out = n % width_col;
    int64_t index = n / width_col;
    int h_out = index % height_col;
    index = index / height_col;
    int t_out = index % depth_col;
    int channel_in = index / depth_col;
    int channel_out = channel_in * ksize_t * ksize_h * ksize_w;
    int t_in = t_out * stride_t - pad_t;
    int h_in = h_out * stride_h - pad_h;
    int w_in = w_out * stride_w - pad_w;
    auto v = data_vol + ((channel_in * depth + t_in) * height + h_in) * width + w_in;
    auto c = data_col + ((channel_out * depth_col + t_out) * height_col + h_out) * width_col + w_out;
    for (int i = 0; i < ksize_t; ++i) {
      for (int j = 0; j < ksize_h; ++j) {
        for (int k = 0; k < ksize_w; ++k) {
          int t = t_in + i * dilation_t;
          int h = h_in + j * dilation_h;
          int w = w_in + k * dilation_w;
          *c = (t >= 0 && h >= 0 && w >= 0 && t < depth && h < height && w < width)
              ? v[i * dilation_t * height * width +
                  j * dilation_h * width + k * dilation_w]
              : static_cast<T>(0);
          c += depth_col * height_col * width_col;
        }
      }
    }
  }
}

template <typename T, typename accT>
__global__ void col2vol_kernel(
    const int64_t n,
    const T* data_col,
    const int depth,
    const int height,
    const int width,
    const int kernel_t,
    const int kernel_h,
    const int kernel_w,
    const int pad_t,
    const int pad_h,
    const int pad_w,
    const int stride_t,
    const int stride_h,
    const int stride_w,
    const int dilation_t,
    const int dilation_h,
    const int dilation_w,
    const int depth_col,
    const int height_col,
    const int width_col,
    T* data_vol)
{
  for (int64_t index = blockDim.x * blockIdx.x + threadIdx.x;
                index < n; index += blockDim.x * gridDim.x) {
    accT val = static_cast<accT>(0);
    const int w_im = index % width + pad_w;
    const int h_im = (index / width) % height + pad_h;
    const int t_im = (index / width / height) % depth + pad_t;
    const int c_im = index / (width * height * depth);
    int kernel_extent_w = (kernel_w - 1) * dilation_w + 1;
    int kernel_extent_h = (kernel_h - 1) * dilation_h + 1;
    int kernel_extent_t = (kernel_t - 1) * dilation_t + 1;
    // compute the start and end of the output
    const int w_col_start =
        (w_im < kernel_extent_w) ? 0 : (w_im - kernel_extent_w) / stride_w + 1;
    const int w_col_end = min(w_im / stride_w + 1, width_col);
    const int h_col_start =
        (h_im < kernel_extent_h) ? 0 : (h_im - kernel_extent_h) / stride_h + 1;
    const int h_col_end = min(h_im / stride_h + 1, height_col);
    const int t_col_start =
        (t_im < kernel_extent_t) ? 0 : (t_im - kernel_extent_t) / stride_t + 1;
    const int t_col_end = min(t_im / stride_t + 1, depth_col);
    // TODO: use LCM of stride and dilation to avoid unnecessary loops
    for (int t_col = t_col_start; t_col < t_col_end; t_col += 1) {
      for (int h_col = h_col_start; h_col < h_col_end; h_col += 1) {
        for (int w_col = w_col_start; w_col < w_col_end; w_col += 1) {
          int64_t t_k = (t_im - t_col * stride_t);
          int64_t h_k = (h_im - h_col * stride_h);
          int64_t w_k = (w_im - w_col * stride_w);
          if (t_k % dilation_t == 0 && h_k % dilation_h == 0 &&
              w_k % dilation_w == 0) {
            t_k /= dilation_t;
            h_k /= dilation_h;
            w_k /= dilation_w;
            const int64_t idx_k =
                ((c_im * kernel_t + t_k) * kernel_h + h_k) * kernel_w + w_k;
            const int64_t data_col_index =
                ((idx_k * depth_col + t_col) *
                    height_col + h_col) *
                  width_col + w_col;
            val += data_col[data_col_index];
          }
        }
      }
    }
    data_vol[index] = static_cast<T>(val);
  }
}

int get_blocks (int64_t n) {
  int64_t numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;
  hipDeviceProp_t devProp;
  hipGetDeviceProperties(&devProp, 0);
  int64_t blocksPerGrid = std::min((int64_t)devProp.maxGridSize[0], numBlocks);
  return blocksPerGrid;
}

template <typename T>
void eval (
    const int repeat,
    const int channels,
    const int depth,
    const int height,
    const int width,
    const int depth_col,
    const int height_col,
    const int width_col,
    const int ksize_t,
    const int ksize_h,
    const int ksize_w,
    const int pad_t,
    const int pad_h,
    const int pad_w,
    const int stride_t,
    const int stride_h,
    const int stride_w,
    const int dilation_t,
    const int dilation_h,
    const int dilation_w)
{
  int64_t vol_size = (int64_t) channels * (2*pad_t+depth) * (2*pad_h+height) * (2*pad_w+width);
  int64_t col_size = ((int64_t) channels * ksize_t * ksize_h * ksize_w + 1) *
                    (depth_col+pad_t) * (height_col+pad_h) * (width_col+pad_w);

  int64_t vol_size_bytes = sizeof(T) * vol_size;
  int64_t col_size_bytes = sizeof(T) * col_size;

  T *h_data_vol = (T*) malloc (vol_size_bytes);
  T *h_data_col = (T*) malloc (col_size_bytes);

  T *h_data_vol_ref = (T*) malloc (vol_size_bytes);
  T *h_data_col_ref = (T*) malloc (col_size_bytes);

  srand(123);
  for (int64_t i = 0; i < vol_size; i++) {
    h_data_vol_ref[i] = h_data_vol[i] = rand() / (T) RAND_MAX;
  }

  T *d_data_vol;
  hipMalloc((void**)&d_data_vol, vol_size_bytes);
  hipMemcpy(d_data_vol, h_data_vol, vol_size_bytes, hipMemcpyHostToDevice);

  T *d_data_col;
  hipMalloc((void**)&d_data_col, col_size_bytes);
  hipMemset(d_data_col, 0, col_size_bytes);

  // each of "channels * depth_col * height_col * width_col"
  // blocks responsible for copying a single-channel grid.
  // We cast an operand to int64 so that the product will not overflow
  int64_t n = static_cast<int64_t>(channels) * depth_col * height_col * width_col;

  int blocksPerGrid = get_blocks(n);

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    vol2col_kernel<T><<<blocksPerGrid, threadsPerBlock>>>(
      n,
      d_data_vol,
      depth, height, width,
      ksize_t, ksize_h, ksize_w,
      pad_t, pad_h, pad_w,
      stride_t, stride_h, stride_w,
      dilation_t, dilation_h, dilation_w,
      depth_col, height_col, width_col,
      d_data_col);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of vol2col kernel: %f (us)\n", (time * 1e-3f) / repeat);

  hipMemcpy(h_data_col, d_data_col, col_size_bytes, hipMemcpyDeviceToHost);

  // verify
  vol2col_reference<T>(
      h_data_vol,
      channels, depth, height, width,
      ksize_t, ksize_h, ksize_w,
      pad_t, pad_h, pad_w,
      stride_t, stride_h, stride_w,
      dilation_t, dilation_h, dilation_w,
      depth_col, height_col, width_col,
      h_data_col_ref);

  int error = memcmp(h_data_col_ref, h_data_col, col_size_bytes);
  printf("%s\n", error ? "FAIL" : "PASS");

  hipDeviceSynchronize();
  start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    col2vol_kernel<T, T><<<blocksPerGrid, threadsPerBlock>>>(
      n,
      d_data_col,
      depth, height, width,
      ksize_t, ksize_h, ksize_w,
      pad_t, pad_h, pad_w,
      stride_t, stride_h, stride_w,
      dilation_t, dilation_h, dilation_w,
      depth_col, height_col, width_col,
      d_data_vol);
  }

  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of col2vol kernel: %f (us)\n", (time * 1e-3f) / repeat);

  hipMemcpy(h_data_vol, d_data_vol, vol_size_bytes, hipMemcpyDeviceToHost);

  // verify
  col2vol_reference<T, T>(
      h_data_col_ref,
      channels, depth, height, width,
      ksize_t, ksize_h, ksize_w,
      pad_t, pad_h, pad_w,
      stride_t, stride_h, stride_w,
      dilation_t, dilation_h, dilation_w,
      depth_col, height_col, width_col,
      h_data_vol_ref);

  for (int64_t i = 0; i < vol_size; i++) {
    if (std::fabs(h_data_vol_ref[i] - h_data_vol[i]) > 1e-3f) {
      error = 1;
      break;
    }
  }
  printf("%s\n", error ? "FAIL" : "PASS");

  hipFree(d_data_vol);
  hipFree(d_data_col);
  free(h_data_vol);
  free(h_data_col);
  free(h_data_vol_ref);
  free(h_data_col_ref);
}

int main(int argc, char* argv[])
{
  if (argc != 2) {
    printf("Usage: %s <repeat>\n", argv[0]);
    return 1;
  }
  const int repeat = atoi(argv[1]);

  int channels = 4;
  int depth = 3;
  int height = 255;
  int width = 255;
  int pad_t = 1;
  int pad_h = 1;
  int pad_w = 1;
  int stride_t = 2;
  int stride_h = 2;
  int stride_w = 2;
  int dilation_t = 2;
  int dilation_h = 2;
  int dilation_w = 2;
  int depth_col = 3;
  int height_col = 255;
  int width_col = 255;

  for (int k = 1; k <= 9; k = k + 2) {
    printf("\nkernel size: %d\n", k);
    int ksize_t = k;
    int ksize_h = k;
    int ksize_w = k;

    eval<float> (repeat,
                 channels, depth, height, width,
                 depth_col, height_col, width_col,
                 ksize_t, ksize_h, ksize_w,
                 pad_t, pad_h, pad_w,
                 stride_t, stride_h, stride_w,
                 dilation_t, dilation_h, dilation_w);
  }

  return 0;
}
