#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <chrono>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include "reference.h"

/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// width is hidden_dim and height is seq_len
__global__ void gelu_bias_loop(__half* src, const __half* bias, int width, int height)
{
  int x     = blockIdx.x;  // seq length
  int y     = threadIdx.x * 2;
  int batch = blockIdx.y;

  if (x < height) {
    int    index = batch * width * height + x * width;
    for (; y < width; y = y + blockDim.x * 2) {
      auto v_bias = ((half2*)bias)[y >> 1];
      auto v_src  = ((half2*)src)[(index + y) >> 1];
      auto v      = __hadd2(v_src, v_bias);
      auto t      = __half22float2(v);
      t.x    = (0.5f * t.x * (1.0f + tanhf(0.79788456f * (t.x + 0.044715f * t.x * t.x * t.x))));
      t.y    = (0.5f * t.y * (1.0f + tanhf(0.79788456f * (t.y + 0.044715f * t.y * t.y * t.y))));
      ((half2*)src)[(index + y) >> 1] = __float22half2_rn(t);
    }
  }
}

__global__ void gelu_bias_loop_base(__half* src, const __half* bias, int width, int height)
{
  int x     = blockIdx.x;  // seq length
  int batch = blockIdx.y;

  if (x < height) {
    int   index = batch * width * height + x * width;
    for (int y = threadIdx.x; y < width; y = y + blockDim.x) {
      auto v_bias = bias[y];
      auto v_src  = src[index + y];
      auto v      = v_src + v_bias;
      auto t      = __half2float(v);
      t      = (0.5f * t * (1.0f + tanhf(0.79788456f * (t + 0.044715f * t * t * t))));
      src[index + y] = __float2half_rn(t);
    }
  }
}

int main(int argc, char* argv[])
{
  if (argc != 5) {
    printf("Usage: %s <batch> <sequence length> <hidden dimension> <repeat>\n", argv[0]);
    printf("The hidden dimension is a multiple of two\n");
    return 1;
  }

  const int batch_size = atoi(argv[1]);
  const int seq_len = atoi(argv[2]);
  const int hidden_dim = atoi(argv[3]);
  const int repeat = atoi(argv[4]);

  const size_t src_size = (size_t)batch_size * seq_len * hidden_dim;

  const size_t src_size_bytes =  src_size * sizeof(__half);
  const int bias_size_bytes = hidden_dim * sizeof(__half);

  srand(123);
  __half* input = (__half*) malloc (src_size_bytes);
  __half* output = (__half*) malloc (src_size_bytes);
  __half* output_ref = (__half*) malloc (src_size_bytes);
  for (size_t i = 0; i < src_size; i++) {
    output_ref[i] = input[i] = __float2half(rand() / (float)RAND_MAX);
  }

  __half* bias = (__half*) malloc (bias_size_bytes);
  for (int i = 0; i < hidden_dim; i++) {
    bias[i] = __float2half(-6 + (rand() % 12)); 
  }

  __half* d_output;
  hipMalloc((void**)&d_output, src_size_bytes);

  __half* d_bias;
  hipMalloc((void**)&d_bias, bias_size_bytes);
  hipMemcpy(d_bias, bias, bias_size_bytes, hipMemcpyHostToDevice);
  
  int block_size;
  if (hidden_dim >= 4096)
    block_size = 512;
  else if (hidden_dim >= 2048)
    block_size = 256;
  else
    block_size = 128;
 
  dim3 block(block_size, 1);
  dim3 grid(seq_len, batch_size);

  // warmup and verify
  gelu_bias_loop_cpu (output_ref, bias, batch_size, hidden_dim, seq_len);

  hipMemcpy(d_output, input, src_size_bytes, hipMemcpyHostToDevice);
  gelu_bias_loop_base <<<grid, block>>> (d_output, d_bias, hidden_dim, seq_len);
  hipMemcpy(output, d_output, src_size_bytes, hipMemcpyDeviceToHost);

  bool ok = true;
  for (size_t i = 0; i < src_size; i++) {
    if (fabsf(__half2float(output_ref[i]) - __half2float(output[i])) > 1e-3f) {
      ok = false;
      break;
    }
  }
  printf("%s\n", ok ? "PASS" : "FAIL");

  hipMemcpy(d_output, input, src_size_bytes, hipMemcpyHostToDevice);
  gelu_bias_loop <<<grid, block>>> (d_output, d_bias, hidden_dim, seq_len);
  hipMemcpy(output, d_output, src_size_bytes, hipMemcpyDeviceToHost);

  ok = true;
  for (size_t i = 0; i < src_size; i++) {
    if (fabsf(__half2float(output_ref[i]) - __half2float(output[i])) > 1e-3f) {
      ok = false;
      break;
    }
  }
  printf("%s\n", ok ? "PASS" : "FAIL");

  hipDeviceSynchronize();
  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    gelu_bias_loop <<<grid, block>>> (d_output, d_bias, hidden_dim, seq_len);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of vectorized kernel %f (ms)\n", (time * 1e-6f) / repeat);

  start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    gelu_bias_loop_base <<<grid, block>>> (d_output, d_bias, hidden_dim, seq_len);
  }

  hipDeviceSynchronize();
  end = std::chrono::steady_clock::now();
  time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of baseline kernel %f (ms)\n", (time * 1e-6f) / repeat);

  hipFree(d_output);
  hipFree(d_bias);
  free(input);
  free(output);
  free(output_ref);
  free(bias);

  return 0;
}
