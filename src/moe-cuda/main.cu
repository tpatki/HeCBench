#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include "kernels.h"
#include "reference.h"

void topk_softmax(int num_tokens, int num_experts, int topk, int repeat)
{
  const int index_size = num_tokens * topk;
  const int index_size_bytes = index_size * sizeof(int);

  const int weight_size = num_tokens * topk;
  const int weight_size_bytes = weight_size * sizeof(float);

  const int output_size = num_tokens * num_experts;
  const int output_size_bytes = output_size * sizeof(float);

  float *topk_weights = (float*) malloc (weight_size_bytes);
  int *topk_indices = (int*) malloc (index_size_bytes);
  int *token_expert_indices = (int*) malloc (index_size_bytes);
  float *gating_output = (float*) malloc (output_size_bytes);

  float *softmax_workspace = (float*) malloc (output_size_bytes);
  float *topk_weights_ref = (float*) malloc (weight_size_bytes);
  int *topk_indices_ref = (int*) malloc (index_size_bytes);
  int *token_expert_indices_ref = (int*) malloc (index_size_bytes);

  srand(123);
  for (int i = 0; i < output_size; i++) {
    gating_output[i] = rand() % 20; 
  }
  for (int i = 0; i < topk; i++) {
    for (int j = 0; j < num_tokens; j++) {
      topk_indices[i * num_tokens + j] = rand() % num_experts;   
    }
  }

  moeSoftmax_reference(
          gating_output,
          nullptr,
          softmax_workspace,
          num_tokens,
          num_experts);

  moeTopK_reference(
          softmax_workspace,
          nullptr,
          topk_weights_ref,
          topk_indices_ref,
          token_expert_indices_ref,
          num_tokens,
          num_experts,
          topk,
          0,  // start_expert
          num_experts);

  float *d_topk_weights;
  hipMalloc(&d_topk_weights, weight_size_bytes);

  int *d_topk_indices;
  hipMalloc(&d_topk_indices, index_size_bytes);

  int *d_token_expert_indices;
  hipMalloc(&d_token_expert_indices, index_size_bytes);

  float *d_gating_output;
  hipMalloc(&d_gating_output, output_size_bytes);

  hipMemcpy(d_gating_output, gating_output, output_size_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_topk_indices, topk_indices, index_size_bytes, hipMemcpyHostToDevice);

  float *d_softmax_workspace;
  hipMalloc(&d_softmax_workspace, output_size_bytes);

  static constexpr int TPB = 256;

  moeSoftmax<TPB><<<num_tokens, TPB>>>(
          d_gating_output,
          nullptr,
          d_softmax_workspace,
          num_experts);

  moeTopK<TPB><<<num_tokens, TPB>>>(
          d_softmax_workspace,
          nullptr,
          d_topk_weights,
          d_topk_indices,
          d_token_expert_indices,
          num_experts,
          topk,
          0,  // start_expert
          num_experts);

  hipMemcpy(topk_weights, d_topk_weights, weight_size_bytes, hipMemcpyDeviceToHost);
  hipMemcpy(topk_indices, d_topk_indices, index_size_bytes, hipMemcpyDeviceToHost);
  hipMemcpy(token_expert_indices, d_token_expert_indices, index_size_bytes, hipMemcpyDeviceToHost);

  int error = memcmp(topk_indices, topk_indices_ref, index_size_bytes);
  error += memcmp(token_expert_indices, token_expert_indices_ref, index_size_bytes);
  for (int i = 0; i < weight_size; i++) {
    if (fabsf(topk_weights[i] - topk_weights_ref[i]) > 1e-3f)  {
      error = 1;
      break;
    }
  }
  printf("%s\n", error ? "FAIL" : "PASS");

  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    moeSoftmax<TPB><<<num_tokens, TPB>>>(
            d_gating_output,
            nullptr,
            d_softmax_workspace,
            num_experts);

    moeTopK<TPB><<<num_tokens, TPB>>>(
            d_softmax_workspace,
            nullptr,
            d_topk_weights,
            d_topk_indices,
            d_token_expert_indices,
            num_experts,
            topk,
            0,  // start_expert
            num_experts);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of kernels: %f (us)\n", (time * 1e-3f) / repeat);

  hipFree(d_topk_weights);
  hipFree(d_topk_indices);
  hipFree(d_token_expert_indices);
  hipFree(d_gating_output);
  hipFree(d_softmax_workspace);

  free(topk_weights);
  free(topk_indices);
  free(token_expert_indices);
  free(gating_output);

  free(topk_weights_ref);
  free(topk_indices_ref);
  free(token_expert_indices_ref);
  free(softmax_workspace);
}

int main(int argc, char* argv[])
{
  if (argc != 5) {
    printf("Usage: %s <number of tokens> <number of experts> <top K> <repeat>\n", argv[0]);
    return 1;
  }
  const int num_tokens = atoi(argv[1]);
  const int num_experts = atoi(argv[2]);
  const int topk = atoi(argv[3]);
  const int repeat = atoi(argv[4]);
  topk_softmax(num_tokens, num_experts, topk, repeat);
  return 0;
}
