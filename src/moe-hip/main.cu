#include "hip/hip_runtime.h"
#include <chrono>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include "kernels.h"
#include "reference.h"

void topk_softmax(int num_tokens, int num_experts, int topk, int repeat)
{
  const int index_size = num_tokens * topk;
  const int index_size_bytes = index_size * sizeof(int);

  const int weight_size = num_tokens * topk;
  const int weight_size_bytes = weight_size * sizeof(float);

  const int output_size = num_tokens * num_experts;
  const int output_size_bytes = output_size * sizeof(float);

  float *topk_weights = (float*) malloc (weight_size_bytes);
  int *topk_indices = (int*) malloc (index_size_bytes);
  int *token_expert_indices = (int*) malloc (index_size_bytes);
  float *gating_output = (float*) malloc (output_size_bytes);

  float *softmax_workspace = (float*) malloc (output_size_bytes);
  float *topk_weights_ref = (float*) malloc (weight_size_bytes);
  int *topk_indices_ref = (int*) malloc (index_size_bytes);
  int *token_expert_indices_ref = (int*) malloc (index_size_bytes);

  srand(123);
  for (int i = 0; i < output_size; i++) {
    gating_output[i] = rand() % 20; 
  }
  for (int i = 0; i < topk; i++) {
    for (int j = 0; j < num_tokens; j++) {
      topk_indices[i * num_tokens + j] = rand() % num_experts;   
    }
  }

  moeSoftmax_reference(
          gating_output,
          nullptr,
          softmax_workspace,
          num_tokens,
          num_experts);

  moeTopK_reference(
          softmax_workspace,
          nullptr,
          topk_weights_ref,
          topk_indices_ref,
          token_expert_indices_ref,
          num_tokens,
          num_experts,
          topk,
          0,  // start_expert
          num_experts);

  float *d_topk_weights;
  hipMalloc(&d_topk_weights, weight_size_bytes);

  int *d_topk_indices;
  hipMalloc(&d_topk_indices, index_size_bytes);

  int *d_token_expert_indices;
  hipMalloc(&d_token_expert_indices, index_size_bytes);

  float *d_gating_output;
  hipMalloc(&d_gating_output, output_size_bytes);

  hipMemcpy(d_gating_output, gating_output, output_size_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_topk_indices, topk_indices, index_size_bytes, hipMemcpyHostToDevice);

  float *d_softmax_workspace;
  hipMalloc(&d_softmax_workspace, output_size_bytes);

  static constexpr int TPB = 256;

  moeSoftmax<TPB><<<num_tokens, TPB>>>(
          d_gating_output,
          nullptr,
          d_softmax_workspace,
          num_experts);

  moeTopK<TPB><<<num_tokens, TPB>>>(
          d_softmax_workspace,
          nullptr,
          d_topk_weights,
          d_topk_indices,
          d_token_expert_indices,
          num_experts,
          topk,
          0,  // start_expert
          num_experts);

  hipMemcpy(topk_weights, d_topk_weights, weight_size_bytes, hipMemcpyDeviceToHost);
  hipMemcpy(topk_indices, d_topk_indices, index_size_bytes, hipMemcpyDeviceToHost);
  hipMemcpy(token_expert_indices, d_token_expert_indices, index_size_bytes, hipMemcpyDeviceToHost);

  int error;
  error = memcmp(topk_indices, topk_indices_ref, index_size_bytes);
  error += memcmp(token_expert_indices, token_expert_indices_ref, index_size_bytes);
  for (int i = 0; i < weight_size; i++) {
    if (fabsf(topk_weights[i] - topk_weights_ref[i]) > 1e-3f)  {
      error = 1;
      break;
    }
  }
  printf("%s\n", error ? "FAIL" : "PASS");

  auto start = std::chrono::steady_clock::now();

  for (int i = 0; i < repeat; i++) {
    moeSoftmax<TPB><<<num_tokens, TPB>>>(
            d_gating_output,
            nullptr,
            d_softmax_workspace,
            num_experts);

    moeTopK<TPB><<<num_tokens, TPB>>>(
            d_softmax_workspace,
            nullptr,
            d_topk_weights,
            d_topk_indices,
            d_token_expert_indices,
            num_experts,
            topk,
            0,  // start_expert
            num_experts);
  }

  hipDeviceSynchronize();
  auto end = std::chrono::steady_clock::now();
  auto time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  printf("Average execution time of kernels: %f (us)\n", (time * 1e-3f) / repeat);

  hipFree(d_topk_weights);
  hipFree(d_topk_indices);
  hipFree(d_token_expert_indices);
  hipFree(d_gating_output);
  hipFree(d_softmax_workspace);

  free(topk_weights);
  free(topk_indices);
  free(token_expert_indices);
  free(gating_output);

  free(topk_weights_ref);
  free(topk_indices_ref);
  free(token_expert_indices_ref);
  free(softmax_workspace);
}

int main(int argc, char* argv[])
{
  if (argc != 5) {
    printf("Usage: %s <number of tokens> <number of experts> <top K> <repeat>\n", argv[0]);
    return 1;
  }
  const int num_tokens = atoi(argv[1]);
  const int num_experts = atoi(argv[2]);
  const int topk = atoi(argv[3]);
  const int repeat = atoi(argv[4]);
  topk_softmax(num_tokens, num_experts, topk, repeat);
  return 0;
}
